#include "hip/hip_runtime.h"
__global__ void compute_output_gradients(
    const float* d_y,        // [N][C], softmax output
    const int* d_labels,     // [N],   true labels
    const float* d_hL,       // [N][out_dim_L], last layer output (already head-averaged)
    const float* d_wo,       // [C][out_dim_L], output linear W
    float* grad_d_wo,        // [C][out_dim_L], output: grad for W_o
    float* grad_d_hL,        // [N][out_dim_L], output: grad for h_i^(L)
    int N, int C, int out_dim_L)
{
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= N) return;

    // Step 1: compute dL/dz = y_hat - y
    float dL_dz[C];
    for (int c = 0; c < C; ++c) {
        dL_dz[c] = d_y[node*C + c] - (c == d_labels[node] ? 1.0f : 0.0f);
    }

    // Step 2: accumulate grad for W_o: dL/dWo += dL/dz * h_i^(L)^T
    for (int c = 0; c < C; ++c) {
        for (int d = 0; d < out_dim_L; ++d) {
            atomicAdd(&grad_d_wo[c*out_dim_L + d], dL_dz[c] * d_hL[node*out_dim_L + d]);
        }
    }

    // Step 3: backprop to h^(L): dL/dh^(L) = W_o^T * dL/dz, store for next layer
    for (int d = 0; d < out_dim_L; ++d) {
        float sum = 0;
        for (int c = 0; c < C; ++c) {
            sum += d_wo[c*out_dim_L + d] * dL_dz[c];
        }
        grad_d_hL[node*out_dim_L + d] = sum;
    }
}

//.........................................................................................................//

__global__ void gatv2_layer_backward(
    int N, int in_dim, int out_dim, int num_heads,
    const int* d_row_ptr, const int* d_col_idx,
    const float* d_x,         // [N][in_dim]
    const float* d_higher,    // [N][num_heads][out_dim]
    const float* attn_coeff,  // [N][num_heads][max_degree]
    const float* attn_score,  // [N][num_heads][max_degree]
    const float* d_leakyrelu, // [N][num_heads][max_degree][out_dim]
    const float* d_w,         // [num_heads][out_dim][2*in_dim]
    const float* d_a,         // [num_heads][out_dim]
    const float* d_s,         // [N][num_heads][max_degree][out_dim]
    float* grad_w,            // [num_heads][out_dim][2*in_dim]
    float* grad_a,            // [num_heads][out_dim]
    float* grad_x_lower,      // [N][in_dim]
    float negative_slope,
    int max_degree
) {
    int i = blockIdx.x;    // Node index
    int h = blockIdx.y;    // Head index
    int tid = threadIdx.x; // Neighbor offset

    int row_start = d_row_ptr[i];
    int row_end = d_row_ptr[i + 1];
    int deg = row_end - row_start;
    //create a shared memory equal to max_degree
    extern __shared__ float shared_memory[]; //in this i will store dL_d_alpha_ij
    if (tid >= deg) return;

    int jj = row_start + tid;
    int j = d_col_idx[jj];

    // --- Step D.2: dL/d alpha_ij ---
    float dL_d_alpha_ij = 0.0f;
    for (int od = 0; od < out_dim; ++od) {
        float dL_d_h = d_higher[(i * num_heads + h) * out_dim + od];
        for (int id = 0; id < in_dim; ++id) {
            float w_ = d_w[(h*out_dim*2*in_dim) + (od*2*in_dim) + (in_dim + id)];
            dL_d_alpha_ij += dL_d_h * w_ * d_x[j*in_dim + id];
        }
    }
    // Store dL/d alpha_ij in shared memory for later use
    shared_memory[tid] = dL_d_alpha_ij;
    __syncthreads(); // Ensure all threads have written their values


    // --- Step D.4: grad_W direct ---
    float alpha = attn_coeff[(i * num_heads + h) * max_degree + (tid)];
    for (int od = 0; od < out_dim; ++od) {
        float dL_d_h = d_higher[(i * num_heads + h) * out_dim + od];
        for (int id = 0; id < in_dim; ++id) {
            float x_j = d_x[j*in_dim + id];
            atomicAdd(&grad_w[(h*out_dim*2*in_dim) + (od*2*in_dim) + (in_dim + id)], alpha * dL_d_h * x_j);    //here massive sequential addition possible
        }
    }

    // --- Step E.1: dL/d e_ij ---  
    float dL_d_e_ij = 0.0f;
    float alpha_ij = attn_coeff[(i * num_heads + h) * max_degree + tid];
    for (int kk = 0; kk < deg; ++kk) {
        int k = d_col_idx[row_start + kk];
        float alpha_ik = attn_coeff[(i * num_heads + h) * max_degree + kk];
        // For dL_d_alpha_ik, may require shared or global memory,
        // or recompute in another pass for full parallel safety
        float dL_d_alpha_ik = shared_memory[kk];
        dL_d_e_ij += dL_d_alpha_ik * alpha_ik * ((j == k ? 1.0f : 0.0f) - alpha_ij);
    }

    // --- Step E.2: grad_a ---
    int leaky_base = (((i * num_heads + h) * max_degree) + tid) * out_dim;
    for (int od = 0; od < out_dim; ++od) {
        float leaky_val = d_leakyrelu[leaky_base + od];
        float a_contrib = dL_d_e_ij * leaky_val;
        atomicAdd(&grad_a[h * out_dim + od], a_contrib);    // it can make slow, massive sequential addition
    }

    // --- Step E.3: grad_W via attention ---
    float* s_ij = &d_s[leaky_base]; // each [out_dim]
    float leaky_grad_val;
    for (int od = 0; od < out_dim; ++od) {
        leaky_grad_val = (s_ij[od] > 0) ? 1.0f : negative_slope;
        float elem = d_a[h * out_dim + od] * leaky_grad_val * dL_d_e_ij;
        for (int id = 0; id < 2 * in_dim; ++id) {
            float x_concat = (id < in_dim) ? d_x[i*in_dim + id] : d_x[j*in_dim + (id-in_dim)];
            float grad_contrib = elem * x_concat;
            atomicAdd(&grad_w[h * out_dim * 2 * in_dim + od * 2 * in_dim + id], grad_contrib);
        }
    }

   // For node i as a neighbor of node j, accumulate to grad_x_lower[i][*] as per direct formula
    int offset = -1;
    for (int t = d_row_ptr[j]; t < d_row_ptr[j+1]; ++t) {
        if (d_col_idx[t] == i) {
            offset = t - d_row_ptr[j];
            break;
        }
    }

    float alpha_jh_i = attn_coeff[(j * num_heads + h) * max_degree + offset];
    for (int od = 0; od < out_dim; ++od) {
        float dL_d_hj = d_higher[(j * num_heads + h) * out_dim + od];
        // Right-part of W (W_right: [out_dim][in_dim]), maps neighbor features
        for (int id = 0; id < in_dim; ++id) {
            float W_right = d_w[(h * out_dim * 2 * in_dim) + (od * 2 * in_dim) + (in_dim + id)];
            // atomic add: node i, feature id
            atomicAdd(&grad_x_lower[i * in_dim + id],
                    alpha_jh_i * W_right * dL_d_hj);
        }
    }

    // ---- INDIRECT GRADIENT FOR x_i ----
    for (int od = 0; od < out_dim; ++od) {
        // Compute LeakyReLU' for this dimension
        float leaky_grad = (s_ij[od] > 0) ? 1.0f : negative_slope;
        // Compute elementwise product with a
        float elem = d_a[h * out_dim + od] * leaky_grad * dL_d_e_ij;

        // Left part of W for this head/output: [out_dim][in_dim]
        for (int id = 0; id < in_dim; ++id) {
            float W_left = d_w[(h * out_dim * 2 * in_dim) + (od * 2 * in_dim) + id];
            float grad_contrib = elem * W_left;
            atomicAdd(&grad_x_lower[i * in_dim + id], grad_contrib);
        }
    }

}
